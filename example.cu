
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel definition
__global__ void AddTwoVectors(float A[], float B[], float C[]) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    printf("kkkkkkkkkkkkkkkkkkkk");
    int N = 1000; // Size of the vectors
    float A[N], B[N], C[N]; // Arrays for vectors A, B, and C

    // Initialize vectors A and B
    for (int i = 0; i < N; ++i) {
        A[i] = 1;
        B[i] = 3;
    }

    float *d_A, *d_B, *d_C; // Device pointers for vectors A, B, and C

    // Allocate memory on the device for vectors A, B, and C
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_C, N * sizeof(float));
    printf("\nkk2kkkkkkkkkkkkkkkkkk");

    // Copy vectors A and B from host to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);
    printf("\nkk3kkkkkkkkkkkkkkkkkk");

    // Kernel invocation with N threads
    AddTwoVectors<<<1, N>>>(d_A, d_B, d_C);
    printf("\nkk4kkkkkkkkkkkkkkkkkk");

    // Check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    
    // Waits untill all CUDA threads are executed
    hipDeviceSynchronize();
    
    // Copy vector C from device to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}