#include "hip/hip_runtime.h"
#include "tensor.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define THREADS_PER_BLOCK 128

__host__ void cpu_to_cuda(Tensor* tensor) {
    
    float* data_tmp;
    hipMalloc((void **)&data_tmp, tensor->size * sizeof(float));
    hipMemcpy(data_tmp, tensor->data, tensor->size * sizeof(float), hipMemcpyHostToDevice);

    tensor->data = data_tmp;

    const char* device_str = "cuda";
    tensor->device = (char*)malloc(strlen(device_str) + 1);
    strcpy(tensor->device, device_str); 

    printf("Successfully sent tensor to: %s\n", tensor->device);
}

__host__ void cuda_to_cpu(Tensor* tensor) {
    float* data_tmp = (float*)malloc(tensor->size * sizeof(float));

    hipMemcpy(data_tmp, tensor->data, tensor->size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(tensor->data);

    tensor->data = data_tmp;

    const char* device_str = "cpu";
    tensor->device = (char*)malloc(strlen(device_str) + 1);
    strcpy(tensor->device, device_str); 

    printf("Successfully sent tensor to: %s\n", tensor->device);
}

__global__ void add_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data1[i] + data2[i];
    }
}

__host__ void add_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int number_of_blocks = (tensor1->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    add_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, tensor1->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void sub_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size) {
   
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data1[i] - data2[i];
    }
}

__host__ void sub_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int number_of_blocks = (tensor1->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    sub_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, tensor1->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void elementwise_mul_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = data1[i] * data2[i];
    }
}

__host__ void elementwise_mul_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int number_of_blocks = (tensor1->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    elementwise_mul_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor1->data, tensor2->data, result_data, tensor1->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void matmul_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int rows1, int cols1, int cols2) {    

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows1 && col < cols2) {
        float sum = 0.0;
        for (int k = 0; k < cols1; k++) {
            sum += data1[row * cols1 + k] * data2[k * cols2 + col];
        }
        result_data[row * cols2 + col] = sum;
    }

}

__host__ void matmul_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data) {
    
    int rows1 = tensor1->shape[0];
    int cols1 = tensor1->shape[1];
    int cols2 = tensor2->shape[1];

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols2 + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows1 + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matmul_tensor_cuda_kernel<<<numBlocks, threadsPerBlock>>>(tensor1->data, tensor2->data, result_data, rows1, cols1, cols2);


    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}

__global__ void pow_tensor_cuda_kernel(float* data, float power, float* result_data, int size) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        result_data[i] = powf(data[i], power);
    }
}

__host__ void pow_tensor_cuda(Tensor* tensor, float power, float* result_data) {
    
    int number_of_blocks = (tensor->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    pow_tensor_cuda_kernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(tensor->data, power, result_data, tensor->size);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipDeviceSynchronize();
}


